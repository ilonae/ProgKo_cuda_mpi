#include <iostream>
#include <cstdlib>
#include <stdio.h>
#include "hip/hip_runtime.h"


__global__ void addVectors(const int *a, const int *b, int *c, const int &size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    c[i] = a[i] + b[i];
    printf("hello from gpu");
};

int main()
{
    const int size = 1024;
    int *a = new int[size];
    int *b = new int[size];
    int *c = new int[size];

    for (int i = 0; i < size; i++)
    {
        a[i] = 1;
        b[i] = 2;
    }

    int *dev_a;
    int *dev_b;
    int *dev_c;

    hipMalloc(&dev_a, sizeof(int) * size);
    hipMalloc(&dev_b, sizeof(int) * size);
    hipMalloc(&dev_c, sizeof(int) * size);

    hipMemcpy(dev_a, a, sizeof(int) * size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, sizeof(int) * size, hipMemcpyHostToDevice);

    int blockCount = 1;
    int blockSize = size;
    addVectors<<<4, size / 4>>>(dev_a, dev_b, dev_c, size);

    hipMemcpy(c, dev_c, sizeof(int) * size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipFree(&dev_a);
    hipFree(&dev_b);
    hipFree(&dev_c);
    hipDeviceReset();

    float avg = 0;
    for (int i = 0; i < size; i++)
        avg += c[i];
    avg /= size;
    std::cout << "Average is: " << avg << ", should be: 3.0" << std::endl;
    delete[] a;
    delete[] b;
    delete[] c;
    return 0;
}