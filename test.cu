#include <iostream>
#include <cstdlib>
#include <stdio.h>
#include "hip/hip_runtime.h"


__global__ void addVectors(const int *a, const int *b, int *c, const int &size)
{
    for (int i = 0; i < size; i++)
    {
        c[i] = a[i] + b[i];
    }
}

int main()
{
    const int size = 1024;

    int *a = new int[size];
    int *b = new int[size];
    int *c = new int[size];

    for (int i = 0; i < size; i++)
    {
        a[i] = 1;
        b[i] = 2;
    }

    addVectors<<<1, 1>>>(a, b, c, size);

    float avg = 0;
    for (int i = 0; i < size; i++)
        avg += c[i];
    avg /= size;
    std::cout << "Average is: " << avg << ", should be: 3.0" << std::endl;
    delete[] a;
    delete[] b;
    delete[] c;
    return 0;
}