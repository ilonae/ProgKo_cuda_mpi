#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

int *dev_a;
int *dev_b;
int *dev_c;

hipMalloc(&dev_a, sizeof(int) * size);
hipMalloc(&dev_b, sizeof(int) * size);
hipMalloc(&dev_c, sizeof(int) * size);

hipMemcpy(dev_a, a, sizeof(int) * size, hipMemcpyHostToDevice);
hipMemcpy(dev_b, b, sizeof(int) * size, hipMemcpyHostToDevice);

__global__ void addVectors(const int *a, const int *b, int *c, const int &size)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int blockCount = 1;
int blockSize = size;
addVectors<<<blockCount, blockSize>>>(dev_a, dev_b, dev_c, size);

hipMemcpy(c, dev_c, sizeof(int) * size, hipMemcpyDeviceToHost);

hipFree(&dev_a);
hipFree(&dev_b);
hipFree(&dev_c);
hipDeviceReset();