#include <iostream>
#include <cstdlib>
#include <stdio.h>
#include "hip/hip_runtime.h"


__global__ void addVectors(const int *a, const int *b, int *c, const int &size)
{
    int i = blockIdx.x * blockDim.x * blockDim.y +
            blockDim.x * threadIdx.y +
            threadIdx.x;
    c[i] = a[i] + b[i];
    printf("hello from gpu");
};

int main()
{
    hipError_t status;
    const int size = 1024;
    int *a = new int[size];
    int *b = new int[size];
    int *c = new int[size];

    for (int i = 0; i < size; i++)
    {
        a[i] = 1;
        b[i] = 2;
    }

    int *dev_a;
    int *dev_b;
    int *dev_c;
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    for (int device = 0; device < deviceCount; ++device)
        {
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, device);
            std::cout << "Device '" << deviceProp.name << "' (" << device << ") has compute capability " << deviceProp.major << "." << deviceProp.minor << " and " << deviceProp.totalGlobalMem << " Bytes of available memory" << std::endl;
        }

    hipMalloc(&dev_a, sizeof(int) * size);
    status = hipMalloc(&dev_a, sizeof(float) * size);
    if (status != hipSuccess)
    {
        std::cerr << "Error with memory allocation!" << std::endl;
        return status;
    }
    hipMalloc(&dev_b, sizeof(int) * size);
    hipMalloc(&dev_c, sizeof(int) * size);

    status = hipMemcpy(dev_a, a, sizeof(float) * size, hipMemcpyHostToDevice);
    if (status != hipSuccess)
    {
        std::cerr << "Error with data copying!" << std::endl;
        return status;
    }
    hipMemcpy(dev_b, b, sizeof(int) * size, hipMemcpyHostToDevice);
    dim3 gridSize(16);
    dim3 blockSize(8, 8);
    addVectors<<<gridSize, blockSize>>>(dev_a, dev_b, dev_c, size);
    status = hipGetLastError();
    if (status != hipSuccess)
    {
        std::cerr << "Error with the kernel!" << std::endl;
        return status;
    }

    hipMemcpy(c, dev_c, sizeof(int) * size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipFree(&dev_a);
    hipFree(&dev_b);
    hipFree(&dev_c);
    hipDeviceReset();

    float avg = 0;
    for (int i = 0; i < size; i++)
        avg += c[i];
    avg /= size;
    std::cout << "Average is: " << avg << ", should be: 3.0" << std::endl;
    delete[] a;
    delete[] b;
    delete[] c;
    return 0;
}