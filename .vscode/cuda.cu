#include "hip/hip_runtime.h"
#include "Header.h"
#include <iostream>
#include <fstream>

//https://github.com/evlasblom/cuda-opencv-examples/blob/master/src/bgrtogray.cu



__global__ void grayscale_kernel(unsigned char* input, unsigned char* output, int* width, int* height, int* colorWidthStep, int* grayWidthStep) {
    {
        const int* x = blockIdx.x * blockDim.x + threadIdx.x;
        const int* y = blockIdx.y * blockDim.y + threadIdx.y;
        if ((x < width) && (y < height))
        {
            //Loc base Image
            const int* color_tid = y * colorWidthStep + (4 * x);

            //Loc in Grayscale
            const int* gray_tid = y * colorWidthStep + (4 * x);

            const unsigned char blue = input[color_tid];
            const unsigned char green = input[color_tid + 1];
            const unsigned char red = input[color_tid + 2];
            const unsigned char alpha = input[color_tid + 3];
            const float gray = red * 0.21f + green * 0.72 + blue * 0.07f;

            output[gray_tid] = static_cast<unsigned char>(gray);
            output[gray_tid+1] = static_cast<unsigned char>(gray);
            output[gray_tid+2] = static_cast<unsigned char>(gray);
            output[gray_tid+3] = static_cast<unsigned char>(alpha);
        }
    }

}


__global__ void emboss_kernel(unsigned char* input, unsigned char* output, int width, int height, int colorWidthStep, int grayWidthStep) {

    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int xminus = blockIdx.x * blockDim.x + (threadIdx.x-1);
    const int yminus = blockIdx.y * blockDim.y + (threadIdx.y-1);
                                    
    if ((x < width) && (y < height)&&(xminus>0)&&(yminus>0))
    {
        //Loc base Image
        const int color_tid = y * colorWidthStep + (4 * x);
        // 100 *4 + 4 *100
        //     40 + 40
        //Loc in Grayscale
        const int emboss_tid = (yminus * colorWidthStep) + (4 * xminus);
        //99*4+4*99
        // 36+36

        //
        //
        const float RGB[3]{ input[color_tid] * 1.0f, input[color_tid + 1] * 1.0f, input[color_tid + 2] * 1.0f };

        const float RGBdiff[3]{ input[emboss_tid] * 1.0f, input[emboss_tid + 1] * 1.0f, input[emboss_tid + 2] * 1.0f };
            
        
        const float diffs[3]{ RGB[0] - RGBdiff[0], RGB[1] - RGBdiff[1], RGB[2] - RGBdiff[2] };


        float diff = diffs[0];
        if (abs(diffs[1])>abs(diff)) { diff = diffs[1]; }
        if (abs(diffs[2]) > abs(diff)) { diff = diffs[2]; }

        float gray = 128 + diff;
        if (gray > 255) { gray = 255; }
        if (gray < 0) { gray = 0; }
        output[color_tid] = static_cast<unsigned char>(gray);
        output[color_tid + 1] = static_cast<unsigned char>(gray);
        output[color_tid + 2] = static_cast<unsigned char>(gray);
        const unsigned char alpha = input[color_tid + 3];
        output[color_tid + 3] = static_cast<unsigned char>(alpha);
    }
}
    
    
    
    
    
int main(int argc, char **argv)
{

    for (int i = 0; i < argc; ++i) 
    std::cout << argv[i] << "\n";
    
    unsigned char* inp=(unsigned char*)argv[1];
    unsigned char* out=(unsigned char*)argv[2];

    int* pw=(int*)argv[3];
    int* ph=(int*)argv[4];

    int* colorwidthstep=(int*)argv[5];
    int* grayWidthStep=(int*)argv[6];

   
    dim3 gridSize(16);
    dim3 blockSize(8, 8);
    
    grayscale_kernel<<<gridSize, blockSize>>>(inp, out,pw,ph,colorwidthstep, grayWidthStep);
}
    
    
    
    
    
    
    ////row
     //const int x = blockIdx.x * blockDim.x + threadIdx.x;
     ////column
     //const int y = blockIdx.y * blockDim.y + threadIdx.y;
     //const int color_tid = y * colorWidthStep + (4 * x);
     //
   //if ((x < width) && (y < height))
    //{
     //   int c = 0;
     //   for (int i = -1; i < 2; i++) {
     //       for (int j = -1; j < 2; j++) {
     //                         
     //           kernel[c] = (x + i) * colorWidthStep + ( (x + i));
     //           //printf("KErnelvalue %d", kernel[c]);
     //           c++;                
     //       }
     //   }
     //   int filterarray[9] = { -1,0,0,0,0,0,0,0,1 };
     //   int j=0;
     //   float calc = 0;
     //   for (int i=0; i < 36;i++) {
     //       if (i % 4 == 0) {
     //           j++;
     //      }
     //      calc=calc+ filterarray[j] * input[kernel[i]];
     //      
     //      printf("calc %d \n", kernel[c]);
     //   }
     //   output[color_tid] = static_cast<unsigned char>(calc);
     //   output[color_tid + 1] = static_cast<unsigned char>(calc);
     //   output[color_tid + 2] = static_cast<unsigned char>(calc);
     //   const unsigned char alpha = input[color_tid + 3];
     //   output[color_tid + 3] = static_cast<unsigned char>(alpha);
     //
    //
    //}


//}
    



/* void convert(const cv::Mat& input, cv::Mat& output,bool flag) {
    // Calculate total number of bytes of input and output image
    const int colorBytes = input.step * input.rows;
    const int grayBytes = output.step * output.rows;
    
    unsigned int* d_kernel;

    unsigned char* d_input, * d_output;

    // Allocate device memory
    SAFE_CALL(hipMalloc<unsigned char>(&d_input, colorBytes), "CUDA Malloc Failed");
    SAFE_CALL(hipMalloc<unsigned char>(&d_output, grayBytes), "CUDA Malloc Failed");    
    // Copy data from OpenCV input image to device memory
    SAFE_CALL(hipMemcpy(d_input, input.ptr(), colorBytes, hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");

    // Threads per Block
    const dim3 block(32, 32);

    // Calculate grid size to cover the whole image
    const dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);

    // Launch the color conversion kernel
    if(flag ==true){
    grayscale_kernel << <grid, block >> > (d_input, d_output, input.cols, input.rows, input.step, output.step);
    }else {    
    emboss_kernel << <grid, block >> > (d_input, d_output, input.cols, input.rows, input.step, output.step);
    }
    // Synchronize to check for any kernel launch errors
    SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");

    // Copy back data from destination device meory to OpenCV output image
    SAFE_CALL(hipMemcpy(output.ptr(), d_output, grayBytes, hipMemcpyDeviceToHost), "CUDA Memcpy Host To Device Failed");

    // Free the device memory
    SAFE_CALL(hipFree(d_input), "CUDA Free Failed");
    SAFE_CALL(hipFree(d_output), "CUDA Free Failed");
} */
