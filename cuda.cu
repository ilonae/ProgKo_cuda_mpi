#include "hip/hip_runtime.h"
#include "Header.h"

//https://github.com/evlasblom/cuda-opencv-examples/blob/master/src/bgrtogray.cu
#include <unistd.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <stdarg.h>

#define PNG_DEBUG 3
#include <png.h>

void abort_(const char * s, ...)
{
        va_list args;
        va_start(args, s);
        vfprintf(stderr, s, args);
        fprintf(stderr, "\n");
        va_end(args);
        abort();
}

int x, y;

int width, height;
png_byte color_type;
png_byte bit_depth;

png_structp png_ptr;
png_structp output_ptr;
png_infop info_ptr;
int number_of_passes;
png_bytep * row_pointers;




__global__ void grayscale_kernel(unsigned char* output, int width, int height, png_bytep * row_pointers) {
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;
        if ((x < width) && (y < height))
        {

            for (y=0; y<height; y++) {
                png_byte* row = row_pointers[y];
                for (x=0; x<width; x++) {
                        png_byte* ptr = &(row[x*3]);
                        printf("Pixel at position [ %d - %d ] has RGB values: %d - %d - %d \n",
                               x, y, ptr[0], ptr[1], ptr[2]);

                               ptr[0],ptr[1],ptr[2] = (ptr[0] + ptr[1] + ptr[2])/3;

                }
        }
            //Loc base Image
            /* const int color_tid = y * colorWidthStep + (4 * x);

            //Loc in Grayscale
            const int gray_tid = y * colorWidthStep + (4 * x);

            const unsigned char blue = input[color_tid];
            const unsigned char green = input[color_tid + 1];
            const unsigned char red = input[color_tid + 2];
            const unsigned char alpha = input[color_tid + 3];
            const float gray = red * 0.21f + green * 0.72 + blue * 0.07f;

            output[gray_tid] = static_cast<unsigned char>(gray);
            output[gray_tid+1] = static_cast<unsigned char>(gray);
            output[gray_tid+2] = static_cast<unsigned char>(gray);
            output[gray_tid+3] = static_cast<unsigned char>(alpha); */
        }
    }

}


__global__ void emboss_kernel(unsigned char* input, unsigned char* output, int width, int height, int colorWidthStep, int grayWidthStep) {

    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int xminus = blockIdx.x * blockDim.x + (threadIdx.x-1);
    const int yminus = blockIdx.y * blockDim.y + (threadIdx.y-1);
                                    
    if ((x < width) && (y < height)&&(xminus>0)&&(yminus>0))
    {
        //Loc base Image
        const int color_tid = y * colorWidthStep + (4 * x);
        // 100 *4 + 4 *100
        //     40 + 40
        //Loc in Grayscale
        const int emboss_tid = (yminus * colorWidthStep) + (4 * xminus);
        //99*4+4*99
        // 36+36

        //
        //
        const float RGB[3]{ input[color_tid] * 1.0f, input[color_tid + 1] * 1.0f, input[color_tid + 2] * 1.0f };

        const float RGBdiff[3]{ input[emboss_tid] * 1.0f, input[emboss_tid + 1] * 1.0f, input[emboss_tid + 2] * 1.0f };
            
        
        const float diffs[3]{ RGB[0] - RGBdiff[0], RGB[1] - RGBdiff[1], RGB[2] - RGBdiff[2] };


        float diff = diffs[0];
        if (abs(diffs[1])>abs(diff)) { diff = diffs[1]; }
        if (abs(diffs[2]) > abs(diff)) { diff = diffs[2]; }

        float gray = 128 + diff;
        if (gray > 255) { gray = 255; }
        if (gray < 0) { gray = 0; }
        output[color_tid] = static_cast<unsigned char>(gray);
        output[color_tid + 1] = static_cast<unsigned char>(gray);
        output[color_tid + 2] = static_cast<unsigned char>(gray);
        const unsigned char alpha = input[color_tid + 3];
        output[color_tid + 3] = static_cast<unsigned char>(alpha);
    }
}

void readpng_version_info()
{
    fprintf(stderr, "   Compiled with libpng %s; using libpng %s.\n",
      PNG_LIBPNG_VER_STRING, png_libpng_ver);
    fprintf(stderr, "   Compiled with zlib %s; using zlib %s.\n",
      ZLIB_VERSION, zlib_version);
}

void read_png_file(char* file_name)
{
        char header[8];

        /* open file and test for it being a png */
        FILE *fp = fopen(file_name, "rb");
        if (!fp)
                abort_("[read_png_file] File %s could not be opened for reading", file_name);
        fread(header, 1, 8, fp);
        

        /* initialize stuff */
        png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);

        if (!png_ptr)
                abort_("[read_png_file] png_create_read_struct failed");

        info_ptr = png_create_info_struct(png_ptr);
        if (!info_ptr)
                abort_("[read_png_file] png_create_info_struct failed");

        if (setjmp(png_jmpbuf(png_ptr)))
                abort_("[read_png_file] Error during init_io");

        png_init_io(png_ptr, fp);
        png_set_sig_bytes(png_ptr, 8);

        png_read_info(png_ptr, info_ptr);

        width = png_get_image_width(png_ptr, info_ptr);
        height = png_get_image_height(png_ptr, info_ptr);
        color_type = png_get_color_type(png_ptr, info_ptr);
        bit_depth = png_get_bit_depth(png_ptr, info_ptr);

        number_of_passes = png_set_interlace_handling(png_ptr);
        png_read_update_info(png_ptr, info_ptr);


        /* read file */
        if (setjmp(png_jmpbuf(png_ptr)))
                abort_("[read_png_file] Error during read_image");

        row_pointers = (png_bytep*) malloc(sizeof(png_bytep) * height);
        for (y=0; y<height; y++)
                row_pointers[y] = (png_byte*) malloc(png_get_rowbytes(png_ptr,info_ptr));

        png_read_image(png_ptr, row_pointers);
        std::cout << "FIle processed.";

        fclose(fp);
}

void process_file(void)
{
        if (png_get_color_type(png_ptr, info_ptr) != PNG_COLOR_TYPE_RGB)
                abort_("[process_file] input file is PNG_COLOR_TYPE_RGBA but must be PNG_COLOR_TYPE_RGB "
                       "(lacks the alpha channel)");

        if (png_get_color_type(png_ptr, info_ptr) != PNG_COLOR_TYPE_RGB)
                abort_("[process_file] color_type of input file must be PNG_COLOR_TYPE_RGB (%d) (is %d)",
                       PNG_COLOR_TYPE_RGB, png_get_color_type(png_ptr, info_ptr));

        int colorBytes =  width * height * 3;


        unsigned char* d_input, * d_output;
        int grayBytes = colorBytes;
        bool flag= true;

        output_ptr = png_ptr;
    
        // Allocate device memory
        SAFE_CALL(hipMalloc<unsigned char>(&d_input, colorBytes), "CUDA Malloc Failed");
        SAFE_CALL(hipMalloc<unsigned char>(&d_output, grayBytes), "CUDA Malloc Failed");    
        // Copy data from OpenCV input image to device memory
        SAFE_CALL(hipMemcpy(d_input, png_ptr, colorBytes, hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");
    
        // Threads per Block
        const dim3 block(32, 32);
    
        // Calculate grid size to cover the whole image
        const dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

         // Launch the color conversion kernel
        if(flag ==true){
            grayscale_kernel << <grid, block >> > (d_output, width, height, row_pointers);
            }

            // Synchronize to check for any kernel launch errors
        /* SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");

        // Copy back data from destination device meory to OpenCV output image
        SAFE_CALL(hipMemcpy(output_ptr, d_output, grayBytes, hipMemcpyDeviceToHost), "CUDA Memcpy Host To Device Failed");

        // Free the device memory
        SAFE_CALL(hipFree(d_input), "CUDA Free Failed");
        SAFE_CALL(hipFree(d_output), "CUDA Free Failed"); */
    
}
 
    
int main(int argc, char **argv)
{
    readpng_version_info();
    read_png_file(argv[1]);
    process_file();
    return 0;
}


//}
    
/* void convert(const cv::Mat& input, cv::Mat& output,bool flag) {
    // Calculate total number of bytes of input and output image
    const int colorBytes = input.step * input.rows;
    const int grayBytes = output.step * output.rows;
    
    unsigned int* d_kernel;

    unsigned char* d_input, * d_output;

    // Allocate device memory
    SAFE_CALL(hipMalloc<unsigned char>(&d_input, colorBytes), "CUDA Malloc Failed");
    SAFE_CALL(hipMalloc<unsigned char>(&d_output, grayBytes), "CUDA Malloc Failed");    
    // Copy data from OpenCV input image to device memory
    SAFE_CALL(hipMemcpy(d_input, input.ptr(), colorBytes, hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");

    // Threads per Block
    const dim3 block(32, 32);

    // Calculate grid size to cover the whole image
    const dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);

    // Launch the color conversion kernel
    if(flag ==true){
    grayscale_kernel << <grid, block >> > (d_input, d_output, input.cols, input.rows, input.step, output.step);
    }else {    
    emboss_kernel << <grid, block >> > (d_input, d_output, input.cols, input.rows, input.step, output.step);
    }
    // Synchronize to check for any kernel launch errors
    SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");

    // Copy back data from destination device meory to OpenCV output image
    SAFE_CALL(hipMemcpy(output.ptr(), d_output, grayBytes, hipMemcpyDeviceToHost), "CUDA Memcpy Host To Device Failed");

    // Free the device memory
    SAFE_CALL(hipFree(d_input), "CUDA Free Failed");
    SAFE_CALL(hipFree(d_output), "CUDA Free Failed");
}  */
