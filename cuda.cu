#include "hip/hip_runtime.h"
#include "Header.h"

//https://github.com/evlasblom/cuda-opencv-examples/blob/master/src/bgrtogray.cu
#include <unistd.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <stdarg.h>

#define PNG_DEBUG 3
#include <png.h>

int x, y;

int width, height;
png_byte color_type;
png_byte bit_depth;

png_structp png_ptr;
png_structp output_ptr;
png_infop info_ptr;
int number_of_passes;
png_bytep * row_pointers;


void abort_(const char * s, ...)
{
        va_list args;
        va_start(args, s);
        vfprintf(stderr, s, args);
        fprintf(stderr, "\n");
        va_end(args);
        abort();
}

__global__ void grayscale_kernel(unsigned char* output, int width, int height, png_bytep* row_pointers) {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;
        
        if ((x < width) && (y < height)){
            png_bytep row = row_pointers[y];
            /* png_bytep px = &(row[x * 3]);
            png_byte old[3 * sizeof(png_byte)];
            memcpy(old, px, sizeof(old));
            px[0] = 255 - old[0];
            px[1] = 255 - old[1];
            px[2] = 255 - old[2]; */
            output[x] = 1;
            output[y] = 1;
            printf("hello from gpu");  
        }  
}


void write_png_file(char* file_name){
        /* create file */
        FILE *fp = fopen(file_name, "wb");
        if (!fp)
                abort_("[write_png_file] File %s could not be opened for writing", file_name);

        /* initialize stuff */
        png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);

        if (!png_ptr)
                abort_("[write_png_file] png_create_write_struct failed");

        info_ptr = png_create_info_struct(png_ptr);
        if (!info_ptr)
                abort_("[write_png_file] png_create_info_struct failed");

        if (setjmp(png_jmpbuf(png_ptr)))
                abort_("[write_png_file] Error during init_io");

        png_init_io(png_ptr, fp);


        /* write header */
        if (setjmp(png_jmpbuf(png_ptr)))
                abort_("[write_png_file] Error during writing header");

        png_set_IHDR(png_ptr, info_ptr, width, height,
                     bit_depth, color_type, PNG_INTERLACE_NONE,
                     PNG_COMPRESSION_TYPE_BASE, PNG_FILTER_TYPE_BASE);

        png_write_info(png_ptr, info_ptr);


        /* write bytes */
        if (setjmp(png_jmpbuf(png_ptr)))
                abort_("[write_png_file] Error during writing bytes");

        png_write_image(png_ptr, row_pointers);


        /* end write */
        if (setjmp(png_jmpbuf(png_ptr)))
                abort_("[write_png_file] Error during end of write");

        png_write_end(png_ptr, NULL);

        /* cleanup heap allocation */
        for (y=0; y<height; y++)
                free(row_pointers[y]);
        free(row_pointers);

        fclose(fp);
}



/* 

__global__ void emboss_kernel(unsigned char* input, unsigned char* output, int width, int height, int colorWidthStep, int grayWidthStep) {

    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int xminus = blockIdx.x * blockDim.x + (threadIdx.x-1);
    const int yminus = blockIdx.y * blockDim.y + (threadIdx.y-1);
                                    
    if ((x < width) && (y < height)&&(xminus>0)&&(yminus>0))
    {
        //Loc base Image
        const int color_tid = y * colorWidthStep + (4 * x);
        // 100 *4 + 4 *100
        //     40 + 40
        //Loc in Grayscale
        const int emboss_tid = (yminus * colorWidthStep) + (4 * xminus);
        //99*4+4*99
        // 36+36

        //
        //
        const float RGB[3]{ input[color_tid] * 1.0f, input[color_tid + 1] * 1.0f, input[color_tid + 2] * 1.0f };

        const float RGBdiff[3]{ input[emboss_tid] * 1.0f, input[emboss_tid + 1] * 1.0f, input[emboss_tid + 2] * 1.0f };
            
        
        const float diffs[3]{ RGB[0] - RGBdiff[0], RGB[1] - RGBdiff[1], RGB[2] - RGBdiff[2] };


        float diff = diffs[0];
        if (abs(diffs[1])>abs(diff)) { diff = diffs[1]; }
        if (abs(diffs[2]) > abs(diff)) { diff = diffs[2]; }

        float gray = 128 + diff;
        if (gray > 255) { gray = 255; }
        if (gray < 0) { gray = 0; }
        output[color_tid] = static_cast<unsigned char>(gray);
        output[color_tid + 1] = static_cast<unsigned char>(gray);
        output[color_tid + 2] = static_cast<unsigned char>(gray);
        const unsigned char alpha = input[color_tid + 3];
        output[color_tid + 3] = static_cast<unsigned char>(alpha);
    }
} */

void readpng_version_info()
{
    fprintf(stderr, "   Compiled with libpng %s; using libpng %s.\n",
      PNG_LIBPNG_VER_STRING, png_libpng_ver);
    fprintf(stderr, "   Compiled with zlib %s; using zlib %s.\n",
      ZLIB_VERSION, zlib_version);
}

void read_png_file(char* file_name){
        char header[8];

        FILE *fp = fopen(file_name, "rb");
        if (!fp)
                abort_("[read_png_file] File %s could not be opened for reading", file_name);
        fread(header, 1, 8, fp);

        /* initialize stuff */
        png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);

        if (!png_ptr)
                abort_("[read_png_file] png_create_read_struct failed");

        info_ptr = png_create_info_struct(png_ptr);
        if (!info_ptr)
                abort_("[read_png_file] png_create_info_struct failed");

        if (setjmp(png_jmpbuf(png_ptr)))
                abort_("[read_png_file] Error during init_io");

        png_init_io(png_ptr, fp);
        png_set_sig_bytes(png_ptr, 8);

        png_read_info(png_ptr, info_ptr);

        width = png_get_image_width(png_ptr, info_ptr);
        height = png_get_image_height(png_ptr, info_ptr);
        color_type = png_get_color_type(png_ptr, info_ptr);
        bit_depth = png_get_bit_depth(png_ptr, info_ptr);

        number_of_passes = png_set_interlace_handling(png_ptr);
        png_read_update_info(png_ptr, info_ptr);


        /* read file */
        if (setjmp(png_jmpbuf(png_ptr)))
                abort_("[read_png_file] Error during read_image");

        row_pointers = (png_bytep*) malloc(sizeof(png_bytep) * height);
        for (y=0; y<height; y++)
                row_pointers[y] = (png_byte*) malloc(png_get_rowbytes(png_ptr,info_ptr));

        png_read_image(png_ptr, row_pointers);
        std::cout << "File read.";

        fclose(fp);
}

void process_file(void)
{
        if (png_get_color_type(png_ptr, info_ptr) != PNG_COLOR_TYPE_RGB)
                abort_("[process_file] input file is PNG_COLOR_TYPE_RGBA but must be PNG_COLOR_TYPE_RGB "
                       "(lacks the alpha channel)");

        if (png_get_color_type(png_ptr, info_ptr) != PNG_COLOR_TYPE_RGB)
                abort_("[process_file] color_type of input file must be PNG_COLOR_TYPE_RGB (%d) (is %d)",
                       PNG_COLOR_TYPE_RGB, png_get_color_type(png_ptr, info_ptr));

        int colorBytes =  width * height * 3;


        unsigned char* d_input, * d_output;
        int grayBytes = colorBytes;
        bool flag= true;
        hipError_t status;
        output_ptr = png_ptr;
    
        // Allocate device memory
        SAFE_CALL(hipMalloc<unsigned char>(&d_input, colorBytes), "CUDA Malloc Failed");
        SAFE_CALL(hipMalloc<unsigned char>(&d_output, grayBytes), "CUDA Malloc Failed");    
        // Copy data from OpenCV input image to device memory
        SAFE_CALL(hipMemcpy(d_input, png_ptr, colorBytes, hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");
    
        // Threads per Block
        const dim3 block(32, 32);
    
        // Calculate grid size to cover the whole image
        const dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

         // Launch the color conversion kernel
        if(flag ==true){
            grayscale_kernel<<<grid, block>>>(d_output, width, height, row_pointers);
            std::cout << "File converted.";
            }
        status = hipGetLastError();
        if (status != hipSuccess)
        {
            std::cerr << "Error with the kernel!" << std::endl;
        }

        

            // Synchronize to check for any kernel launch errors
        SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");

        // Copy back data from destination device meory to OpenCV output image
        SAFE_CALL(hipMemcpy(output_ptr, d_output, grayBytes, hipMemcpyDeviceToHost), "CUDA Memcpy Host To Device Failed");

        // Free the device memory
        SAFE_CALL(hipFree(d_input), "CUDA Free Failed");
        SAFE_CALL(hipFree(d_output), "CUDA Free Failed");
        SAFE_CALL(hipDeviceReset(), "Reset Failed");
    
}
 
    
int main(int argc, char **argv)
{
    readpng_version_info();
    read_png_file(argv[1]);
    process_file();
    write_png_file(argv[2]);
    return 0;
}
